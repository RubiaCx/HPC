
#include <hip/hip_runtime.h>
// transfer float4
#define FLOAT4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])

// ElementWise Add
// grid(N/128), block(128)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add(float *a, float *b, float *c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        c[idx] = a[idx] + b[idx];
}

// ElementWise Add + Vec4
// grid(N/128), block(128/4)
// a: Nx1, b: Nx1, c: Nx1, c = elementwise_add(a, b)
__global__ void elementwise_add_vec4(float *a, float *b, float *c, int N)
{
    int idx = 4 * (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < N)
    {
        float4 reg_a = FLOAT4(a[idx]);
        float4 reg_b = FLOAT4(b[idx]);
        float4 reg_c;
        reg_c.x = reg_a.x + reg_b.x;
        reg_c.y = reg_a.y + reg_b.y;
        reg_c.z = reg_a.z + reg_b.z;
        reg_c.w = reg_a.w + reg_b.w;
        FLOAT4(c[idx]) = reg_c;
    }
}